#include "hip/hip_runtime.h"
//Zachary G. Nicolaou 2/4/2024
//Dormand Prince 4/5 stepper on the GPU
#include "dp45.h"

//dp45 coefficients
const float a1loc[1] = {1.0/5};
const float a2loc[2] = {3.0/40, 9.0/40};
const float a3loc[3] = {44.0/45, -56.0/15, 32.0/9};
const float a4loc[4] = {19372.0/6561, -25360.0/2187, 64448.0/6561, -212.0/729};
const float a5loc[5] = {9017.0/3168, -355.0/33, 46732.0/5247, 49.0/176, -5103.0/18656};
const float b1loc[6] = {35.0/384, 0, 500.0/1113, 125.0/192, -2187.0/6784, 11.0/84};
const float c[6] = {0.0, 1.0/5, 3.0/10, 4.0/5, 8.0/9, 1.0};
const float eloc[7] = {-71.0/57600, 0, 71.0/16695, -71.0/1920, 17253.0/339200, -22.0/525,1.0/40};

const float p1loc[4] = {1.0, -8048581381.0/2820520608, 8663915743.0/2820520608, -12715105075.0/11282082432};
const float p2loc[4] = {0, 0, 0, 0};
const float p3loc[4] = {0, 131558114200.0/32700410799, -68118460800.0/10900136933,87487479700.0/32700410799};
const float p4loc[4] = {0, -1754552775.0/470086768, 14199869525.0/1410260304, -10690763975.0/1880347072};
const float p5loc[4] = {0, 127303824393.0/49829197408, -318862633887.0/49829197408, 701980252875.0/199316789632};
const float p6loc[4] = {0, -282668133.0/205662961, 2019193451.0/616988883, -1453857185.0/822651844};
const float p7loc[4] = {0, 40617522.0/29380423, -110615467.0/29380423, 69997945.0/29380423};


static float *y, *ylast, *ytemp, *yerr, *y_eval, *k1, *k2, *k3, *k4, *k5, *k6, *k7;
static float *a1, *a2, *a3, *a4, *a5, *b1, *e, *p1, *p2, *p3, *p4, *p5, *p6, *p7;

static unsigned long int N;
static void (*dydt)(float, float*, float*, void*) = NULL;
static float atl, rtl, t_last;
static int fixed;
static hipblasHandle_t handle;

//Steps for the DP stepper
__global__ void step2 (float* y, float* k1, float* ytemp, const float *a1, const float h, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    ytemp[i]=y[i]+h*a1[0]*k1[i];
  }
}
__global__ void step3 (float* y, float* k1, float* k2, float* ytemp, const float *a2, const float h, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    ytemp[i]=y[i]+h*(a2[0]*k1[i]+a2[1]*k2[i]);
  }
}
__global__ void step4 (float* y, float* k1, float* k2, float* k3, float* ytemp, const float *a3, const float h, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    ytemp[i]=y[i]+h*(a3[0]*k1[i]+a3[1]*k2[i]+a3[2]*k3[i]);
  }
}
__global__ void step5 (float* y, float* k1, float* k2, float* k3, float* k4, float* ytemp, const float *a4, const float h, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    ytemp[i]=y[i]+h*(a4[0]*k1[i]+a4[1]*k2[i]+a4[2]*k3[i]+a4[3]*k4[i]);
  }
}
__global__ void step6 (float* y, float* k1, float* k2, float* k3, float* k4, float* k5, float* ytemp, const float *a5, const float h, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    ytemp[i]=y[i]+h*(a5[0]*k1[i]+a5[1]*k2[i]+a5[2]*k3[i]+a5[3]*k4[i]+a5[4]*k5[i]);
  }
}
__global__ void step7 (float* y, float* k1, float* k2, float* k3, float* k4, float* k5, float* k6, float *ytemp, const float *b1, const float h, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    ytemp[i]=y[i]+h*(b1[0]*k1[i]+b1[1]*k2[i]+b1[2]*k3[i]+b1[3]*k4[i]+b1[4]*k5[i]+b1[5]*k6[i]);
  }
}
__global__ void interpolate (float* y, float* k1, float* k2, float* k3, float* k4, float* k5, float* k6, float* k7, float *ytemp, const float *p1, const float *p2,const float *p3,const float *p4,const float *p5,const float *p6,const float *p7, const float h, const float h2, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    float x=h2/h;
    ytemp[i]=y[i]+h*(x*(p1[0]*k1[i]+p2[0]*k2[i]+p3[0]*k3[i]+p4[0]*k4[i]+p5[0]*k5[i]+p6[0]*k6[i]+p7[0]*k7[i])+x*x*(p1[1]*k1[i]+p2[1]*k2[i]+p3[1]*k3[i]+p4[1]*k4[i]+p5[1]*k5[i]+p6[1]*k6[i]+p7[1]*k7[i])+x*x*x*(p1[2]*k1[i]+p2[2]*k2[i]+p3[2]*k3[i]+p4[2]*k4[i]+p5[2]*k5[i]+p6[2]*k6[i]+p7[2]*k7[i])+x*x*x*x*(p1[3]*k1[i]+p2[3]*k2[i]+p3[3]*k3[i]+p4[3]*k4[i]+p5[3]*k5[i]+p6[3]*k6[i]+p7[3]*k7[i]));
  }
}

//Error estimate for the DP stepper
__global__ void error (float *y, float *ytemp, float* k1, float* k2, float* k3, float* k4, float* k5, float* k6, float *k7, float* yerr, const float *e, const float h, const float atl, const float rtl, const unsigned long int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    yerr[i]=h*(e[0]*k1[i]+e[1]*k2[i]+e[2]*k3[i]+e[3]*k4[i]+e[4]*k5[i]+e[5]*k6[i]+e[6]*k7[i])/(atl+rtl*fmax(fabs(y[i]),fabs(ytemp[i])));
  }
}

//Attempt a DP step
int dp45_step (float *t, float *h, void* pars){
  float norm=0;
  //Calculate the intermediate steps and error estimates using the CUDA kernels
  step2<<<(N+255)/256, 256>>>(y, k1, ytemp, a1, *h, N);
  (*dydt)((*t)+(*h)*c[1],ytemp,k2,pars);

  step3<<<(N+255)/256, 256>>>(y, k1, k2, ytemp, a2, *h, N);
  (*dydt)((*t)+(*h)*c[2],ytemp,k3,pars);

  step4<<<(N+255)/256, 256>>>(y, k1, k2, k3, ytemp, a3, *h, N);
  (*dydt)((*t)+(*h)*c[3],ytemp,k4,pars);

  step5<<<(N+255)/256, 256>>>(y, k1, k2, k3, k4, ytemp, a4, *h, N);
  (*dydt)((*t)+(*h)*c[4],ytemp,k5,pars);

  step6<<<(N+255)/256, 256>>>(y, k1, k2, k3, k4, k5, ytemp, a5, *h, N);
  (*dydt)((*t)+(*h)*c[5],ytemp,k6,pars);

  step7<<<(N+255)/256, 256>>>(y,  k1, k2, k3, k4, k5, k6, ytemp, b1, *h, N);
  (*dydt)((*t)+(*h),ytemp,k7,pars);


  if(fixed){
    t_last=*t;
    hipblasScopy(handle, N, y, 1, ylast, 1);
    hipblasScopy(handle, N, ytemp, 1, y, 1);
    (*t)=(*t)+(*h);
    return 1;
  }
  else{
    error<<<(N+255)/256, 256>>>(y, ytemp, k1, k2, k3, k4, k5, k6, k7, yerr, e, *h, atl, rtl, N);
    hipblasSnrm2(handle, N, yerr, 1, &norm);
    norm/=pow(N,0.5);
    float factor=0.9*pow(norm,-0.2);
    //Accept or reject the step and update the step size
    if(norm<1){
      t_last=*t;
      hipblasScopy(handle, N, y, 1, ylast, 1);
      hipblasScopy(handle, N, ytemp, 1, y, 1);


      (*t)=(*t)+(*h);
      if (factor>10)
        factor=10;
      (*h)*=factor;
      return 1;
    }
    else if (factor<1){
      if (factor<0.2)
        factor=0.2;
      (*h)*=factor;
    }
  }
  return 0;
}

float *dp45_eval(const float t,const float t_eval){
  interpolate<<<(N+255)/256, 256>>>(ylast, k1, k2, k3, k4, k5, k6, k7, y_eval, p1, p2, p3, p4, p5, p6, p7, t-t_last, t_eval-t_last, N);
  return y_eval;
}

float* dp45_run(float *t, float *h, float t1, void *pars, void (*step_eval)(float, float, float*, void*)){

  hipMalloc ((void**)&y_eval, N*sizeof(float));
  (*dydt)(*t,y,k1,pars);

  while(*t<t1){
    // if(*t+*h>t1)
    //   *h=t1-*t;

    int success=dp45_step (t, h, pars);
    if(success){
      (*step_eval)(*t,*h,y,pars);
      hipblasScopy(handle, N, k7, 1, k1, 1);
    }
  }
  return y;
}

float* dp45_init(int n, float atol, float rtol, int fixedstep, float *yloc, hipblasHandle_t h, void (*func)(float, float*, float*, void*)){
  N=n;
  rtl=rtol;
  atl=atol;
  fixed=fixedstep;
  dydt=func;
  handle=h;

  hipMalloc ((void**)&y, N*sizeof(float));
  hipMalloc ((void**)&yerr, N*sizeof(float));
  hipMalloc ((void**)&ytemp, N*sizeof(float));
  hipMalloc ((void**)&ylast, N*sizeof(float));
  hipMalloc ((void**)&k1, N*sizeof(float));
  hipMalloc ((void**)&k2, N*sizeof(float));
  hipMalloc ((void**)&k3, N*sizeof(float));
  hipMalloc ((void**)&k4, N*sizeof(float));
  hipMalloc ((void**)&k5, N*sizeof(float));
  hipMalloc ((void**)&k6, N*sizeof(float));
  hipMalloc ((void**)&k7, N*sizeof(float));

  hipMalloc ((void**)&a1, 1*sizeof(float));
  hipMalloc ((void**)&a2, 2*sizeof(float));
  hipMalloc ((void**)&a3, 3*sizeof(float));
  hipMalloc ((void**)&a4, 4*sizeof(float));
  hipMalloc ((void**)&a5, 5*sizeof(float));
  hipMalloc ((void**)&b1, 6*sizeof(float));
  hipMalloc ((void**)&e, 7*sizeof(float));

  hipMalloc ((void**)&p1, 4*sizeof(float));
  hipMalloc ((void**)&p2, 4*sizeof(float));
  hipMalloc ((void**)&p3, 4*sizeof(float));
  hipMalloc ((void**)&p4, 4*sizeof(float));
  hipMalloc ((void**)&p5, 4*sizeof(float));
  hipMalloc ((void**)&p6, 4*sizeof(float));
  hipMalloc ((void**)&p7, 4*sizeof(float));

  hipMemcpy (y, yloc, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (a1, a1loc, 1*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (a2, a2loc, 2*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (a3, a3loc, 3*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (a4, a4loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (a5, a5loc, 5*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (b1, b1loc, 6*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (e, eloc, 7*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy (p1, p1loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (p2, p2loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (p3, p3loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (p4, p4loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (p5, p5loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (p6, p6loc, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (p7, p7loc, 4*sizeof(float), hipMemcpyHostToDevice);

  return y;
}

void dp45_destroy(){
  hipFree(y);
  hipFree(yerr);
  hipFree(ytemp);
  hipFree(ylast);
  hipFree(y_eval);
  hipFree(k1);
  hipFree(k2);
  hipFree(k3);
  hipFree(k4);
  hipFree(k5);
  hipFree(k6);
  hipFree(k7);
  hipFree(a1);
  hipFree(a2);
  hipFree(a3);
  hipFree(a4);
  hipFree(a5);
  hipFree(b1);
  hipFree(e);
  hipFree(p1);
  hipFree(p2);
  hipFree(p3);
  hipFree(p4);
  hipFree(p5);
  hipFree(p6);
  hipFree(p7);


}
